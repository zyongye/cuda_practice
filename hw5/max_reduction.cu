

#include <hip/hip_runtime.h>
#include <__clang_cuda_runtime_wrapper.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t N = 8ULL*1024ULL*1024ULL;  // data size
const int BLOCK_SIZE = 256;  // CUDA maximum is 1024

__global__ void reduce(float *gdata, float *out, size_t n){
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    sdata[tid] = 0.0f;
    size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

    while (idx < n) {  // grid stride loop to load data
        sdata[tid] = max(gdata[idx], sdata[tid]);
        idx += gridDim.x*blockDim.x;  
    }

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        __syncthreads();
        if (tid < s)  // parallel sweep reduction
            sdata[tid] = max(sdata[tid + s], sdata[tid]);
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}

int main(){

  float *h_A, *h_sum, *d_A, *d_sums;
  const int blocks = 640;
  h_A = new float[N];  // allocate space for data in host memory
  h_sum = new float;
  float max_val = 5.0f;
  for (size_t i = 0; i < N; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  h_A[100] = max_val;
  hipMalloc(&d_A, N*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, blocks*sizeof(float));  // allocate device space for partial sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  //cuda processing sequence step 1 is complete
  reduce<<<blocks, BLOCK_SIZE>>>(d_A, d_sums, N); // reduce stage 1
  cudaCheckErrors("reduction kernel launch failure");
  reduce<<<1, BLOCK_SIZE>>>(d_sums, d_A, blocks); // reduce stage 2
  cudaCheckErrors("reduction kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_A, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction w/atomic kernel execution failure or hipMemcpy D2H failure");
  printf("reduction output: %f, expected sum reduction output: %f, expected max reduction output: %f\n", *h_sum, (float)((N-1)+max_val), max_val);
  return 0;
}

#include <iostream>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    const int m = 512, n = 512, k = 512;  // Larger size to highlight Tensor Core usage

    float *A_fp32 = new float[m * k];
    float *B_fp32 = new float[k * n];
    float *C_fp32 = new float[m * n]();

    for (int i = 0; i < m * k; ++i) A_fp32[i] = 1.0f;
    for (int i = 0; i < k * n; ++i) B_fp32[i] = 1.0f;

    __hip_bfloat16 *A_bf16 = new __hip_bfloat16[m * k];
    __hip_bfloat16 *B_bf16 = new __hip_bfloat16[k * n];
    for (int i = 0; i < m * k; ++i) A_bf16[i] = __float2bfloat16(A_fp32[i]);
    for (int i = 0; i < k * n; ++i) B_bf16[i] = __float2bfloat16(B_fp32[i]);

    __hip_bfloat16 *d_A, *d_B;
    float *d_C;
    CHECK_CUDA(hipMalloc(&d_A, m * k * sizeof(__hip_bfloat16)));
    CHECK_CUDA(hipMalloc(&d_B, k * n * sizeof(__hip_bfloat16)));
    CHECK_CUDA(hipMalloc(&d_C, m * n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, A_bf16, m * k * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B_bf16, k * n * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, C_fp32, m * n * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f, beta = 0.0f;

    // --- Profiling ---
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));

    // Tensor Core GEMM
    CHECK_CUBLAS(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k,
        &alpha,
        d_B, HIP_R_16BF, n,
        d_A, HIP_R_16BF, k,
        &beta,
        d_C, HIP_R_32F, n,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP  // Tensor Core path
    ));

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float elapsed_ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_ms, start, stop));
    std::cout << "Tensor Core BF16 GEMM time: " << elapsed_ms << " ms\n";

    // Optionally copy back and check a few values
    CHECK_CUDA(hipMemcpy(C_fp32, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "C[0] = " << C_fp32[0] << " (should be close to " << k << ")\n";

    // Cleanup
    delete[] A_fp32;
    delete[] B_fp32;
    delete[] C_fp32;
    delete[] A_bf16;
    delete[] B_bf16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
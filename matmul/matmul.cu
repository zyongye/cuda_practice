#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "../util/matmul_ref.h"

#include "./kernels/kernel1.cuh"
#include "./kernels/kernel2.cuh"
#include "./kernels/kernel3.cuh"

#define M 4096
#define N 4096
#define K 4096

int main(){
    float *h_A, *h_B, *h_C, *h_C_ref;
    float *d_A, *d_B, *d_C, *d_C_ref;
    float alpha, beta;
    
    h_A = new float[M * K];
    h_B = new float[K * N];
    h_C = new float[M * N];
    h_C_ref = new float[M * N];

    init_rand(h_A, M * K);
    init_rand(h_B, K * N);
    init_rand(h_C, M * N);
    alpha = (rand() - RAND_MAX / 2 ) / (float)RAND_MAX;;
    beta = (rand() - RAND_MAX / 2 ) / (float)RAND_MAX;;

    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));
    hipMalloc(&d_C_ref, M * N * sizeof(float));
    cudaCheckErrors("Malloc fails");

    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C_ref, h_C, M * N * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memcpy fails");

    sgemm3(d_C, d_A, d_B, alpha, beta, M, N, K);
    cudaCheckErrors("Error executing kernel");

    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy back to host fails");

    printf("Kernel completes\n");

    // running reference implementation
    sgemm2(d_C_ref, d_A, d_B, alpha, beta, M, N, K);
    cudaCheckErrors("Error executing reference kernel");

    hipMemcpy(h_C_ref, d_C_ref, M * N * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy back to host fails");

    if(assert_close(h_C,h_C_ref, M * N, 0.02, 0.004)){
        printf("Success!\n"); 
    }else{
        printf("error!\n");
    }

    return 0;
}






#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const size_t DSIZE = 2048;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024

// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds){

  int idx = blockIdx.x * blockDim.x + threadIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx * ds + i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}

// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = blockIdx.x * blockDim.x + threadIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[i * ds + idx];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}

bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}

int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
    
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
    
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float)); // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
    
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
    
  //cuda processing sequence step 1 is complete
  row_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
    
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums correct!\n");
    
  hipMemset(d_sums, 0, DSIZE*sizeof(float));
    
  column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
    
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("column sums correct!\n");
  return 0;
}
  
